#include "hip/hip_runtime.h"
#ifndef __NVCC__        // defined by nvcc complier 
#define __NVCC__
#endif

#include "Network.h"
#include "Helpers.h"
#include "Parameters.h"

#include <hiprand.h>
#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/common.h>

using namespace tcnn;
using precision_t = tcnn::network_precision_t;

#define GPUMatrix GPUMatrix<float, CM>

namespace {

    // cuda related
    hipStream_t inference_stream;
    hipStream_t training_stream;
    hiprandGenerator_t rng;

    struct _Network { 
        std::shared_ptr<Loss<precision_t>> loss = nullptr;
        std::shared_ptr<Optimizer<precision_t>> optimizer = nullptr;
        std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = nullptr;
        std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer = nullptr;
    };

    struct _Memory {
        // the GPUMatrix class supports MxN matrices only
        // the GPUMatrix store in a continuous area in memory, either row major or column major
        GPUMatrix* training_data = nullptr;
        GPUMatrix* training_target = nullptr;
        GPUMatrix* inference_data = nullptr;
        GPUMatrix* inference_target = nullptr;
        GPUMatrix* training_self_query = nullptr;
        GPUMatrix* training_self_pred = nullptr;
        GPUMemory<float>* random_seq = nullptr;
    };

    _Memory* mMemory;
    _Network* mNetwork;
}

template <typename T>
__device__ void copyQuery(T* data, const NRC::RadianceQuery* query) {
    // use naive copy kernel since memcpy has bad performance on small datas.
    
    data[0] = query->pos.x, data[1] = query->pos.y, data[2] = query->pos.z;
    data[3] = query->dir.x, data[4] = query->dir.y;
#if AUX_INPUTS
    data[5] = query->roughness;
    data[6] = query->normal.x, data[7] = query->normal.y;
    data[8] = query->diffuse.x, data[9] = query->diffuse.y, data[10] = query->diffuse.z;
    data[11] = query->specular.x, data[12] = query->specular.y, data[13] = query->specular.z;
#endif
}

// linear kernels with only x-dim not 1. must be called using linear_kernal()
// blockDim = 128, threadIdx is the index of a thread within a thread block, i.e. in [0, 128)
// reference linear_kernel() for details.
// stride: input dim
template <uint32_t stride, typename T = float>
__global__ void generateBatchSequential(uint32_t n_elements, uint32_t offset, 
    NRC::RadianceQuery* queries, T* data) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + offset < n_elements) {
        uint32_t data_index = i * stride, query_index = i + offset;

        copyQuery(&data[data_index], &queries[query_index]);
    }
}

template <uint32_t stride, typename T = float>
__global__ void generateTrainingDataFromSamples(uint32_t n_elements, uint32_t offset,
    NRC::RadianceSample* samples, NRC::RadianceQuery* self_queries, T* self_query_pred,
    T* training_data, T* training_target, uint32_t* training_sample_counter, uint32_t* self_query_counter, 
    float* random_indices = nullptr) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + offset > n_elements) return;
    int data_index = i * stride, sample_index = i + offset;
    if (random_indices) 
        sample_index = (1 - random_indices[sample_index]) * *training_sample_counter;

    int pred_index = samples[sample_index].idx; // pred_index == -1 if a self-query is not needed.

    if (sample_index < *training_sample_counter) {
        float3 factor = samples[sample_index].a, bias = samples[sample_index].b;
        uint32_t output_index = i * 3;

        copyQuery(&training_data[data_index], &samples[sample_index].query);

        float3 pred_radiance = { 0, 0, 0 };
        if (pred_index >= 0)    // else the sample doesn't contain a self query.
            pred_radiance = { self_query_pred[pred_index * 3], self_query_pred[pred_index * 3 + 1], self_query_pred[pred_index * 3 + 2] };
#if REFLECTANCE_FACT
        float3 reflectance = samples[sample_index].query.diffuse + samples[sample_index].query.specular;
        if (pred_index >= 0)
            // restore self-query from reflectance factorization...
            pred_radiance = pred_radiance * (self_queries[pred_index].diffuse + self_queries[pred_index].specular); 
        float3 radiance = safe_div(pred_radiance * factor + bias, reflectance);
#else
        float3 radiance = pred_radiance * factor + bias;
#endif
        *(float3*)&training_target[output_index] = radiance;
    }
}

template <typename T = float>
__global__ void mapPredRadianceToScreen(uint32_t n_elements, uint32_t width,
    NRC::RadianceQuery* queries, T* data, hipSurfaceObject_t output) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t x = i % width, y = i / width;
    uint32_t index = i * 3;
    float3 radiance = { data[index + 0] , data[index + 1], data[index + 2] };

#if REFLECTANCE_FACT
    //radiance = radiance * (queries[i].diffuse + queries[i].specular);
#endif
    float4 val = { radiance.x, radiance.y, radiance.z, 1.0f };
    surf2Dwrite(val, output, (int)sizeof(float4) * x, y);
}

template <class T>
__global__ void mapPredRadianceToScreen2(NRC::RadianceQuery* queries, T* data, hipSurfaceObject_t output,
    uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uint32_t index = y * width + x;
        uint32_t data_index = index * 3;
        float3 radiance = { data[data_index + 0], data[data_index + 1], data[data_index + 2]};

#if REFLECTANCE_FACT
        //radiance = radiance * (queries[index].diffuse + queries[index].specular);
#endif
        float4 val = { radiance.x, radiance.y, radiance.z, 1.0f };
        surf2Dwrite(val, output, (int)sizeof(float4) * x, y);
    }
}

template <typename T = float>
__global__ void chkNaN(uint32_t n_elements, T* data) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > n_elements) return;
    if (isnan(data[i]) || isinf(data[i])) {
        data[i] = (T)0.f;
    }
}

using namespace NRC::Parameters;

namespace NRC {
    NRCNetwork::NRCNetwork()
    {
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        CUDA_CHECK_THROW(hipStreamCreate(&training_stream));
        //training_stream = inference_stream;

        hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(rng, 7272ULL);
        hiprandSetStream(rng, training_stream);

        initializeNetwork();
    }

    NRCNetwork::~NRCNetwork()
    {
        delete mNetwork;
        delete mMemory;
    }

    void NRCNetwork::initializeNetwork()
    {
        mNetwork = new _Network();
        mMemory = new _Memory();

        //initialize network
        std::ifstream f(config_path);
        tcnn::json config = tcnn::json::parse(f, nullptr, true, true);

        json loss_opts = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts = config.value("network", json::object());
        json encoding_opts = config.value("encoding", json::object());

        mNetwork->loss = std::shared_ptr<Loss<precision_t>>(create_loss<precision_t>(loss_opts) );
        mNetwork->optimizer = std::shared_ptr<Optimizer<precision_t>>(create_optimizer<precision_t>(optimizer_opts));
#if AUX_INPUTS
        mNetwork->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(8, 6, output_dim, encoding_opts, network_opts);
#else
        mNetwork->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(input_dim, 0, output_dim, encoding_opts, network_opts);
#endif
        mNetwork->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(mNetwork->network, mNetwork->optimizer, mNetwork->loss);

        learning_rate = mNetwork->optimizer->learning_rate();
        mMemory->training_data = new GPUMatrix(input_dim, batch_size);
        mMemory->training_target = new GPUMatrix(output_dim, batch_size);
        mMemory->inference_data = new GPUMatrix(input_dim, resolution);
        mMemory->inference_target = new GPUMatrix(output_dim, resolution);
        mMemory->training_self_query = new GPUMatrix(input_dim, self_query_batch_size);
        mMemory->training_self_pred = new GPUMatrix(output_dim, self_query_batch_size);

        mMemory->random_seq = new GPUMemory<float>(n_train_batch * batch_size);
        hiprandGenerateUniform(rng, mMemory->random_seq->data(), n_train_batch * batch_size);
    }

    void NRCNetwork::reset()
    {
        hipStreamSynchronize(training_stream);
        hipStreamSynchronize(inference_stream);
        mNetwork->trainer->initialize_params(seed);
    }

    void NRCNetwork::inference(RadianceQuery* queries, hipSurfaceObject_t output,
        uint32_t width, uint32_t height)
    {
        uint32_t n_elements = width * height;

        // this input generation process takes about ~1ms.
        linear_kernel(generateBatchSequential<input_dim>, 0, inference_stream, n_elements,
            0, queries, mMemory->inference_data->data());
        
        mNetwork->network->inference(inference_stream, *mMemory->inference_data, *mMemory->inference_target);
        //linear_kernel(mapPredRadianceToScreen<float>, 0, inference_stream, n_elements, width, queries, mMemory->inference_target->data(), output);

        dim3 dimBlock(16, 16), dimGrid(div_round_up(width, 16u), div_round_up(height, 16u));
        mapPredRadianceToScreen2<float> <<<dimGrid, dimBlock, 0, inference_stream >>>
            (queries, mMemory->inference_target->data(), output, width, height);

        hipStreamSynchronize(inference_stream);
    }

    void NRCNetwork::train(RadianceQuery* self_queries, uint32_t* self_query_counter,
        RadianceSample* training_samples, uint32_t* training_sample_counter, float& loss)
    {
        // setup change-able parameters
        mNetwork->optimizer->set_learning_rate(learning_rate);

        // self query
        linear_kernel(generateBatchSequential<input_dim>, 0, training_stream, self_query_batch_size,
            0, self_queries, mMemory->training_self_query->data());
        mNetwork->network->inference(training_stream, *mMemory->training_self_query, *mMemory->training_self_pred);

        // training
#if 1   // randomly select 4 training batches over all samples
        hiprandGenerateUniform(rng, mMemory->random_seq->data(), n_train_batch * batch_size);
        for (uint32_t i = 0; i < n_train_batch; i++) {
            linear_kernel(generateTrainingDataFromSamples<input_dim, float>, 0, training_stream, batch_size,
                i * batch_size, training_samples, self_queries, mMemory->training_self_pred->data(),
                mMemory->training_data->data(), mMemory->training_target->data(),
                training_sample_counter, self_query_counter, mMemory->random_seq->data());
            mNetwork->trainer->training_step(training_stream, *mMemory->training_data, *mMemory->training_target, &loss);
        }
#elif 0   // batched training over all samples
        // get sample count...
        // TODO: this costs ~1.5ms. find a better strategy.
        uint32_t sample_count;
        hipMemcpy(&sample_count, training_sample_counter, sizeof(float), hipMemcpyDeviceToHost);

        for (uint32_t i = 0; i + batch_size <= sample_count; i += batch_size) {
            linear_kernel(generateTrainingDataFromSamples<input_dim, float>, 0, training_stream, batch_size,
                i, training_samples, mMemory->training_self_pred->data(),
                mMemory->training_data->data(), mMemory->training_target->data(),
                training_sample_counter, self_query_counter);

            // not here: we check NaNs and INFs in the shader before adding the training samples.
            //linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_data->n_elements(), mMemory->training_data->data());
            //linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_target->n_elements(), mMemory->training_target->data());

            mNetwork->trainer->training_step(training_stream, *mMemory->training_data, *mMemory->training_target, &loss);
            std::cout << "Loss at current step: " << loss << std::endl;
        }
#elif 0 // single batched training
        linear_kernel(generateTrainingDataFromSamples<input_dim, float>, 0, training_stream, batch_size,
            0, training_samples, mMemory->training_self_pred->data(),
            mMemory->training_data->data(), mMemory->training_target->data(),
            training_sample_counter, self_query_counter);

        linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_data->n_elements(), mMemory->training_data->data());
        linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_target->n_elements(), mMemory->training_target->data());

        mNetwork->trainer->training_step(training_stream, *mMemory->training_data, *mMemory->training_target, &loss);
        std::cout << "Loss at current step: " << loss << std::endl;
#endif
        //hipStreamSynchronize(training_stream);
    }
}
